#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdlib>
#include <hipcub/hipcub.hpp>  // or equivalently <cub/device/device_scan.cuh>
#include <random>
#include <vector>

#include "argparse.hpp"
#include "cpp_utils.h"
#include "cuda_utils.h"

struct Args {
  int size;
  int block_size;
};

int parse_args(int argc, char* argv[], Args& args, hipDeviceProp_t& deviceProp) {
  argparse::ArgumentParser program("scan");
  program.add_argument("--size")
      .help("size of the array to scan")
      .scan<'i', int>()
      .default_value(1 << 24)
      .store_into(args.size);
  program.add_argument("--block-size")
      .help("block size")
      .scan<'i', int>()
      .default_value(256)
      .store_into(args.block_size);
  try {
    program.parse_args(argc, argv);
  } catch (const std::exception& err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    return 1;
  }

  printf("Arguments:\n");
  printf("  Size: %d\n", args.size);
  printf("  Block size: %d\n", args.block_size);
  return 0;
}

void print_vector(const char* name, const std::vector<int>& output_data) {
  printf("%s: [", name);
  for (int i = 0; i < output_data.size(); i++) {
    printf("%d ", output_data[i]);
  }
  printf("\n");
}

std::vector<int> generate_input_data(int size) {
  std::default_random_engine generator(786);
  std::uniform_int_distribution<int> distribution(0, 100);
  std::vector<int> input_data(size);
  for (int i = 0; i < size; i++) {
    input_data[i] = distribution(generator);
  }
  // print_vector("Input", input_data);
  return input_data;
}

bool verify_result(std::vector<int>& output_data_cpu, std::vector<int>& output_data_gpu) {
  // print_vector("CPU", output_data_cpu);
  // print_vector("GPU", output_data_gpu);
  return std::equal(output_data_cpu.begin(), output_data_cpu.end(), output_data_gpu.begin());
}

std::vector<int> cpu_inclusive_scan(const std::vector<int>& input_data) {
  Timer timer("inclusive_scan on the CPU");
  std::vector<int> output_data(input_data.size());
  std::inclusive_scan(input_data.begin(), input_data.end(), output_data.begin());
  return output_data;
}

std::vector<int> cub_inclusive_scan(const std::vector<int>& input_data) {
  Timer timer("inclusive_scan on the GPU using the CUB library");
  std::vector<int> output_data_gpu(input_data.size());
  hipStream_t stream;
  cudaCheck(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  hipEvent_t startEvent, stopEvent;
  cudaCheck(hipEventCreate(&startEvent));
  cudaCheck(hipEventCreate(&stopEvent));

  void* d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  int* d_input_data = nullptr;
  cudaCheck(hipMalloc((void**)&d_input_data, input_data.size() * sizeof(int)));
  int* d_output_data = nullptr;
  cudaCheck(hipMalloc((void**)&d_output_data, input_data.size() * sizeof(int)));

  // Copy input data to device
  cudaCheck(hipMemcpyAsync(d_input_data, input_data.data(), input_data.size() * sizeof(int), hipMemcpyHostToDevice,
                            stream));

  cudaCheck(hipEventRecord(startEvent, stream));

  // Determine temporary device storage requirements
  cudaCheck(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_input_data, d_output_data,
                                          input_data.size(), stream));

  // Allocate temporary storage
  cudaCheck(hipMalloc(&d_temp_storage, temp_storage_bytes));

  // Run exclusive prefix sum
  cudaCheck(hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_input_data, d_output_data,
                                          input_data.size(), stream));
  cudaCheck(hipEventRecord(stopEvent, stream));
  cudaCheck(hipEventSynchronize(stopEvent));
  float gpuExecutionTime = 0;
  cudaCheck(hipEventElapsedTime(&gpuExecutionTime, startEvent, stopEvent));
  printf("GPU time taken to perform inclusive scan on GPU using the CUB library: %f ms\n", gpuExecutionTime);
  cudaCheck(hipMemcpyAsync(output_data_gpu.data(), d_output_data, input_data.size() * sizeof(int),
                            hipMemcpyDeviceToHost, stream));

  cudaCheck(hipFree(d_temp_storage));
  cudaCheck(hipFree(d_input_data));
  cudaCheck(hipFree(d_output_data));
  cudaCheck(hipEventDestroy(startEvent));
  cudaCheck(hipEventDestroy(stopEvent));
  cudaCheck(hipStreamDestroy(stream));
  return output_data_gpu;
}

int main(int argc, char* argv[]) {
  Args args;
  hipDeviceProp_t deviceProp = getDeviceProperties(0, true);
  if (parse_args(argc, argv, args, deviceProp) != 0) {
    return 1;
  }

  // Run inclusive scan on CPU
  const std::vector<int> input_data = generate_input_data(args.size);
  std::vector<int> output_data_cpu = cpu_inclusive_scan(input_data);

  // Run inclusive scan on GPU using the CUB library
  std::vector<int> output_data_gpu = cub_inclusive_scan(input_data);

  // Verify results
  bool result = verify_result(output_data_cpu, output_data_gpu);
  printf("Results match: %s\n", result ? "true" : "false");

  return result ? 0 : 1;
}