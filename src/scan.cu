#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdlib>
#include <hipcub/hipcub.hpp>  // or equivalently <cub/device/device_scan.cuh>
#include <random>
#include <vector>

#include "argparse.hpp"
#include "cpp_utils.h"
#include "cuda_utils.h"

struct Args {
  int size;
  int block_size;
};

int parse_args(int argc, char* argv[], Args& args, hipDeviceProp_t& deviceProp) {
  argparse::ArgumentParser program("scan");
  program.add_argument("--size")
      .help("size of the array to scan")
      .scan<'i', int>()
      .default_value(1 << 24)
      .store_into(args.size);
  program.add_argument("--block-size")
      .help("block size")
      .scan<'i', int>()
      .default_value(256)
      .store_into(args.block_size);
  try {
    program.parse_args(argc, argv);
  } catch (const std::exception& err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    return 1;
  }

  printf("Arguments:\n");
  printf("  Size: %d\n", args.size);
  printf("  Block size: %d\n", args.block_size);
  return 0;
}

void print_vector(const char* name, const std::vector<int>& output_data) {
  printf("%s: [", name);
  for (int i = 0; i < output_data.size(); i++) {
    printf("%d ", output_data[i]);
  }
  printf("\n");
}

std::vector<int> generate_input_data(int size) {
  std::default_random_engine generator(786);
  std::uniform_int_distribution<int> distribution(0, 100);
  std::vector<int> input_data(size);
  for (int i = 0; i < size; i++) {
    input_data[i] = distribution(generator);
  }
  // print_vector("Input", input_data);
  return input_data;
}

bool verify_result(std::vector<int>& output_data_cpu, std::vector<int>& output_data_gpu) {
  // print_vector("CPU", output_data_cpu);
  // print_vector("GPU", output_data_gpu);
  return std::equal(output_data_cpu.begin(), output_data_cpu.end(), output_data_gpu.begin());
}

std::vector<int> cpu_inclusive_scan(const std::vector<int>& input_data) {
  Timer timer("inclusive_scan on the CPU");
  std::vector<int> output_data(input_data.size());
  std::inclusive_scan(input_data.begin(), input_data.end(), output_data.begin());
  return output_data;
}

std::vector<int> cub_inclusive_scan(const std::vector<int>& input_data) {
  Timer timer("inclusive_scan on the GPU using the CUB library");
  std::vector<int> output_data_gpu(input_data.size());
  CudaStream streamWrapper;
  hipStream_t stream = streamWrapper.stream;

  auto d_input_data = make_cuda_unique<int>(input_data.size());
  auto d_output_data = make_cuda_unique<int>(input_data.size());

  // Copy input data to device
  cudaCheck(hipMemcpyAsync(d_input_data.get(), input_data.data(), input_data.size() * sizeof(int),
                            hipMemcpyHostToDevice, stream));

  {
    CudaEventRecorder recorder = streamWrapper.record("inclusive_scan on the GPU using the CUB library");

    // Determine temporary device storage requirements
    size_t temp_storage_bytes = 0;
    cudaCheck(hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, d_input_data.get(), d_output_data.get(),
                                            input_data.size(), stream));

    // Allocate temporary storage
    auto d_temp_storage = make_cuda_unique<char>(temp_storage_bytes);

    // Run exclusive prefix sum
    cudaCheck(hipcub::DeviceScan::InclusiveSum(d_temp_storage.get(), temp_storage_bytes, d_input_data.get(),
                                            d_output_data.get(), input_data.size(), stream));
  }

  cudaCheck(hipMemcpyAsync(output_data_gpu.data(), d_output_data.get(), input_data.size() * sizeof(int),
                            hipMemcpyDeviceToHost, stream));
  cudaCheck(hipStreamSynchronize(stream));

  return output_data_gpu;
}

int main(int argc, char* argv[]) {
  Args args;
  hipDeviceProp_t deviceProp = getDeviceProperties(0, true);
  if (parse_args(argc, argv, args, deviceProp) != 0) {
    return 1;
  }

  // Run inclusive scan on CPU
  const std::vector<int> input_data = generate_input_data(args.size);
  std::vector<int> output_data_cpu = cpu_inclusive_scan(input_data);

  // Run inclusive scan on GPU using the CUB library
  std::vector<int> output_data_gpu = cub_inclusive_scan(input_data);

  // Verify results
  bool result = verify_result(output_data_cpu, output_data_gpu);
  printf("Results match: %s\n", result ? "true" : "false");

  return result ? 0 : 1;
}