#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include <cassert>
#include <cstdlib>
#include <hipcub/hipcub.hpp>  // or equivalently <cub/device/device_scan.cuh>
#include <execution>
#include <functional>
#include <random>
#include <ranges>
#include <vector>

#include "argparse.hpp"
#include "cpp_utils.h"
#include "cuda_utils.h"


struct Args {
  int size;
  int block_size;
  bool debug_print;
};


int parse_args(int argc, char* argv[], Args& args, hipDeviceProp_t& deviceProp) {
  argparse::ArgumentParser program("scan");
  std::string kernel_type;
  program.add_argument("--size")
      .help("The size of the array to scan")
      .scan<'i', int>()
      .default_value(1 << 24)
      .store_into(args.size);
  program.add_argument("--block-size")
      .help("The block size")
      .scan<'i', int>()
      .default_value(1024)
      .store_into(args.block_size);
  program.add_argument("--debug-print")
      .help("Whether to print debug information")
      .default_value(false)
      .store_into(args.debug_print);
  try {
    program.parse_args(argc, argv);
  } catch (const std::exception& err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    return 1;
  }

  printf("Arguments:\n");
  printf("  Size: %d\n", args.size);
  printf("  Block size: %d\n", args.block_size);
  return 0;
}


CudaUniquePtr<int> generate_input_data(int size) {
  auto input_data = make_cuda_unique<int>(size, true);
  std::default_random_engine generator(786);
  std::uniform_int_distribution<int> distribution(0, 100);
  for (int i = 0; i < size; i++) {
    input_data.get()[i] = distribution(generator);
  }
  return input_data;
}

bool verify_result(int* output_data_cpu, int* output_data_gpu, size_t size) {
  for (size_t i = 0; i < size; i++) {
    if (output_data_cpu[i] != output_data_gpu[i]) {
      printf("Output data mismatch at index %zu: %d vs %d, diff: %d\n", i, output_data_cpu[i], output_data_gpu[i],
             output_data_cpu[i] - output_data_gpu[i]);
      return false;
    }
  }
  return true;
}

// This runs serially on the CPU
std::vector<int> compact_stream_cpu_serial(int* input_data, size_t size, std::function<bool(int)> predicate) {
  Timer timer("compact_stream_cpu_serial");
  std::vector<int> output_data;
  std::copy_if(input_data, input_data + size, std::back_inserter(output_data), predicate);
  return output_data;
}

// This runs in parallel on the CPU using the STL, which by default uses OpenMP as well, but we could make it run
// on the GPU too by installing the HPC SDK and compiling with nvc++ and stdpar=gpu: nvc++ -std=c++20 -stdpar=gpu -O3
std::vector<int> compact_stream_cpu_parallel_stl(int* input_data, size_t size, std::function<bool(int)> predicate) {
  Timer timer("compact_stream_cpu_parallel_stl");
  // Create a vector of 0 or 1 depending on predicate result
  std::vector<int> output_data_indicators(size);
  std::transform(std::execution::par, input_data, input_data + size, output_data_indicators.begin(),
                 [predicate](int x) { return predicate(x) ? 1 : 0; });

  // Run an inclusive scan, when the sum changes, that's the index of the next element to copy, the last index is the
  // size of the output data
  std::inclusive_scan(std::execution::par, output_data_indicators.begin(), output_data_indicators.end(),
                      output_data_indicators.begin());

  std::vector<int> output_data(output_data_indicators.back());
  auto indexes = std::views::iota((size_t)0, (size_t)size);
  std::for_each(std::execution::par, indexes.begin(), indexes.end(),
                [predicate, input_data, output_data_indicators, &output_data](size_t i) {
                  if (predicate(input_data[i])) {
                    int index = output_data_indicators[i];
                    output_data[index - 1] = input_data[i];
                  }
                });
  return output_data;
}


// This runs in parallel on the CPU using OpenMP.
// TODO - fuse the 3 blocks together and optimize it further.
std::vector<int> compact_stream_cpu_parallel_omp(int* input_data, size_t size, std::function<bool(int)> predicate,
                                                 int block_size) {
  Timer timer("compact_stream_cpu_parallel_omp");
  std::vector<int> output_data_indicators(size);
  std::vector<int> output_data_indicators_prefix_sum(size);
  unsigned int sum = 0;

#pragma omp parallel for schedule(static, block_size) default(shared)
  for (int i = 0; i < size; i++) {
    output_data_indicators[i] = predicate(input_data[i]) ? 1 : 0;
  }

#pragma omp parallel for simd reduction(inscan, + : sum)
  for (int i = 0; i < size; i++) {
    sum += output_data_indicators[i];
#pragma omp scan inclusive(sum)
    output_data_indicators_prefix_sum[i] = sum;
  }

  auto output_size = output_data_indicators_prefix_sum.back();
  std::vector<int> output_data(output_size);

#pragma omp parallel for schedule(static, block_size) default(shared)
  for (int i = 0; i < size; i++) {
    if (predicate(input_data[i])) {
      output_data[output_data_indicators_prefix_sum[i] - 1] = input_data[i];
    }
  }
  return output_data;
}


template <typename Predicate>
__global__ void create_input_data_indicators_kernel(int* input_data, int* input_data_indicators, size_t size,
                                                    Predicate predicate) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < size) {
    input_data_indicators[tid] = predicate(input_data[tid]) ? 1 : 0;
  }
}

template <typename Predicate>
void create_input_data_indicators(CudaStream& streamWrapper, int* input_data, int* input_data_indicators, size_t size,
                                  int block_size, Predicate predicate) {
  auto recorder = streamWrapper.record("create_input_data_indicators_kernel");
  hipStream_t stream = streamWrapper.stream;
  dim3 dimBlock(block_size);
  dim3 dimGrid((size + block_size - 1) / block_size);

  create_input_data_indicators_kernel<<<dimGrid, dimBlock, 0, stream>>>(input_data, input_data_indicators, size,
                                                                        predicate);
  cudaCheck(hipGetLastError());
}

void cub_inclusive_scan(CudaStream& streamWrapper, CudaUniquePtr<int>& d_input_data, CudaUniquePtr<int>& d_output_data,
                        int size) {
  hipStream_t stream = streamWrapper.stream;
  CudaEventRecorder recorder = streamWrapper.record("inclusive scan on the GPU using the CUB library");

  // Determine temporary device storage requirements
  size_t temp_storage_bytes = 0;
  cudaCheck(hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, d_input_data.get(), d_output_data.get(), size,
                                          stream));
  cudaCheck(hipGetLastError());

  // Allocate temporary storage
  auto d_temp_storage = make_cuda_unique<char>(temp_storage_bytes);

  // Run exclusive prefix sum
  cudaCheck(hipcub::DeviceScan::InclusiveSum(d_temp_storage.get(), temp_storage_bytes, d_input_data.get(),
                                          d_output_data.get(), size, stream));
  cudaCheck(hipGetLastError());
}


template <typename Predicate>
__global__ void create_output_data_kernel(int* input_data, int* input_data_indicators, int* output_data,
                                          size_t input_size, Predicate predicate) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < input_size && predicate(input_data[tid])) {
    output_data[input_data_indicators[tid] - 1] = input_data[tid];
  }
}

template <typename Predicate>
void create_output_data(CudaStream& streamWrapper, int* input_data, int* input_data_indicators, int* output_data,
                        size_t input_size, int block_size, Predicate predicate) {
  auto recorder = streamWrapper.record("create_output_data_kernel");
  hipStream_t stream = streamWrapper.stream;
  dim3 dimBlock(block_size);
  dim3 dimGrid((input_size + block_size - 1) / block_size);

  create_output_data_kernel<<<dimGrid, dimBlock, 0, stream>>>(input_data, input_data_indicators, output_data,
                                                              input_size, predicate);
  cudaCheck(hipGetLastError());
}

// This function performs stream compaction on the GPU using a 3-pass approach:
// - The first pass runs a kernel that marks the output data with 0 or 1 depending on the predicate
// - The second pass runs an inclusive scan on the output data of the first pass using the CUB library
// - The third pass runs a kernel that copies the input data to the output data based on the indexes
// and output sizefrom the second pass
// TODO - we need to fuse the 3 kernels, calling CUB directly from the device code
template <typename Predicate>
std::vector<int> compact_stream_gpu(int* input_data, size_t size, Predicate predicate, int block_size) {
  Timer timer("compact_stream_gpu");
  CudaStream streamWrapper;
  hipStream_t stream = streamWrapper.stream;
  auto input_data_indicators = make_cuda_unique<int>(size, true);

  create_input_data_indicators(streamWrapper, input_data, input_data_indicators.get(), size, block_size, predicate);

  cub_inclusive_scan(streamWrapper, input_data_indicators, input_data_indicators, size);
  auto output_size = input_data_indicators.get()[size - 1];
  std::vector<int> output_data(output_size);
  auto d_output_data = make_cuda_unique<int>(output_size, true);

  create_output_data(streamWrapper, input_data, input_data_indicators.get(), d_output_data.get(), size, block_size,
                     predicate);

  cudaCheck(hipMemcpyAsync(output_data.data(), d_output_data.get(), output_size * sizeof(int), hipMemcpyDeviceToHost,
                            stream));
  cudaCheck(hipStreamSynchronize(stream));
  cudaCheck(hipGetLastError());
  cudaCheck(hipDeviceSynchronize());
  return output_data;
}

int main(int argc, char* argv[]) {
  Args args;
  hipDeviceProp_t deviceProp = getDeviceProperties(0, true);
  if (parse_args(argc, argv, args, deviceProp) != 0) {
    return 1;
  }
  auto predicate = [](int x) -> bool { return x % 2 == 0; };
  auto input_data = generate_input_data(args.size);
  auto output_data_cpu_serial = compact_stream_cpu_serial(input_data.get(), args.size, predicate);
  auto output_data_cpu_parallel_stl = compact_stream_cpu_parallel_stl(input_data.get(), args.size, predicate);
  auto output_data_cpu_parallel_omp =
      compact_stream_cpu_parallel_omp(input_data.get(), args.size, predicate, args.block_size);

  if (args.debug_print) {
    print_vector("Input data", input_data.get(), args.size);
    print_vector("Output data CPU serial", output_data_cpu_serial);
    print_vector("Output data CPU parallel STL", output_data_cpu_parallel_stl);
    print_vector("Output data CPU parallel OMP", output_data_cpu_parallel_omp);
  }

  bool result =
      verify_result(output_data_cpu_serial.data(), output_data_cpu_parallel_stl.data(), output_data_cpu_serial.size());
  printf("CPU serial and parallel STL results match: %s\n", result ? "true" : "false");
  result =
      verify_result(output_data_cpu_serial.data(), output_data_cpu_parallel_omp.data(), output_data_cpu_serial.size());
  printf("CPU serial and parallel OMP results match: %s\n", result ? "true" : "false");

  auto predicate_gpu = [] __device__(int x) { return x % 2 == 0; };
  auto output_data_gpu = compact_stream_gpu(input_data.get(), args.size, predicate_gpu, args.block_size);
  if (args.debug_print) {
    print_vector("Output data GPU", output_data_gpu);
  }

  result = verify_result(output_data_cpu_serial.data(), output_data_gpu.data(), output_data_cpu_serial.size());
  printf("GPU results match: %s\n", result ? "true" : "false");

  return result ? 0 : 1;
  return 0;
}