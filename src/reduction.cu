#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cassert>
#include <cstdlib>
#include <random>
#include <vector>

#include "argparse.hpp"
#include "cpp_utils.hpp"
#include "cuda_utils.cuh"

namespace cg = cooperative_groups;

struct Args {
  int size;
  int block_size;
  int kernel_number;
};

inline int nextPow2(int n) {
  if (n == 0)
    return 1;
  return 1 << (32 - __builtin_clz(n - 1));
}

int parse_args(int argc, char* argv[], Args& args, hipDeviceProp_t& deviceProp) {
  argparse::ArgumentParser program("histogram");
  program.add_argument("--size")
      .help("size of the array to reduce")
      .scan<'i', int>()
      .default_value(1 << 24)
      .store_into(args.size);
  program.add_argument("--block-size")
      .help("block size")
      .scan<'i', int>()
      .default_value(256)
      .store_into(args.block_size);
  program.add_argument("--kernel-number")
      .help("kernel number")
      .scan<'i', int>()
      .default_value(2)
      .store_into(args.kernel_number);
  try {
    program.parse_args(argc, argv);
  } catch (const std::exception& err) {
    std::cerr << err.what() << std::endl;
    std::cerr << program;
    return 1;
  }

  if (args.block_size < 32) {
    printf("Setting block size to 32\n");
    args.block_size = 32;
  }

  printf("Setting block size to next power of 2\n");
  args.block_size = nextPow2(args.block_size);

  printf("Arguments:\n");
  printf("  Size: %d\n", args.size);
  printf("  Block size: %d\n", args.block_size);
  printf("  Kernel number: %d\n", args.kernel_number);
  return 0;
}

std::vector<int> generateInputData(int size) {
  std::vector<int> inputData(size);
  std::default_random_engine generator(786);
  std::uniform_int_distribution<int> distribution(0, 100);
  for (int i = 0; i < size; i++) {
    inputData[i] = distribution(generator);
  }
  return inputData;
}

unsigned long long reduceCpu(const std::vector<int>& inputData) {
  Timer timer("reduce on the CPU");
  unsigned long long sum = 0;
  for (int i = 0; i < inputData.size(); i++) {
    sum += inputData[i];
  }
  return sum;
}

// Naive kernel where all threads perform an atomic add, this will not perform well because
// it will have a lot of contention on the atomic add
__global__ void reductionKernel0(int* d_inputData, int size, unsigned long long* d_outputData) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < size) {
    atomicAdd(d_outputData, d_inputData[tid]);
  }
}

// In this kernel, each block reduces twice the block size from the input data, each thread reads
// its natural element and the element one block apart from global to shared memory. Then the
// stride is progressively halved until only one result remains, the one for the first thread,
// which is then written to the output. The cooperative threads API are used to synchronize
// every pass into shared memory.
__global__ void reductionKernel1(int* d_inputData, int size, unsigned long long* d_outputData) {
  auto cta = cg::this_thread_block();      // the thread block group
  extern __shared__ unsigned int sdata[];  // blockDim.x * sizeof(unsigned int)

  // Reduce from global memory to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  unsigned int sum = (i < size) ? d_inputData[i] : 0;

  if (i + blockDim.x < size)
    sum += d_inputData[i + blockDim.x];

  sdata[tid] = sum;
  cta.sync();

  // Then reduce from shared memory
  for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      sdata[tid] = sum = sum + sdata[tid + stride];
    }

    cta.sync();
  }

  // The first thread in the block writes the result to global memory
  if (tid == 0) {
    atomicAdd(d_outputData, sum);
  }
}

/// In this kernel, we use a cooperative thread block and thread tiles of size 32 (warps) to reduce
/// the input data using hardware accelerated reduction for warp tiles. Each thread computes its own
/// sum from global memory by adding values over the stride and then each thread sum is reduced across
/// tiles using the CG reduce() function.
__global__ void reductionKernel2(int* d_inputData, int size, unsigned long long* d_outputData) {
  auto cta = cg::this_thread_block();
  auto warp = cg::tiled_partition<32>(cta);
  int tid = cta.group_index().x * (cta.group_dim().x * 2) + cta.thread_index().x;
  unsigned int thread_sum = (tid < size) ? d_inputData[tid] : 0;

  if (tid + cta.group_dim().x < size) {
    thread_sum += d_inputData[tid + cta.group_dim().x];
  }

  // reduce thread sums across each warp, cg::plus<int> allows cg::reduce() to
  // know it can use hardware acceleration for addition
  unsigned int warp_sum = cg::reduce(warp, thread_sum, cg::plus<unsigned int>());

  if (warp.thread_rank() == 0) {
    atomicAdd(d_outputData, (unsigned long long)warp_sum);
  }
}

void runKernel(int* d_inputData, unsigned long long* d_outputData, Args& args) {
  switch (args.kernel_number) {
    case 0: {
      dim3 dimBlock(args.block_size);
      dim3 dimGrid((args.size + args.block_size - 1) / args.block_size);
      reductionKernel0<<<dimGrid, dimBlock, 0>>>(d_inputData, args.size, d_outputData);
      break;
    }
    case 1: {
      dim3 dimBlock(args.block_size);
      dim3 dimGrid((args.size + args.block_size * 2 - 1) / (args.block_size * 2));
      unsigned int shared_mem_size = dimBlock.x * sizeof(unsigned int);
      reductionKernel1<<<dimGrid, dimBlock, shared_mem_size>>>(d_inputData, args.size, d_outputData);
      break;
    }
    case 2: {
      dim3 dimBlock(args.block_size);
      dim3 dimGrid((args.size + args.block_size * 2 - 1) / (args.block_size * 2));
      reductionKernel2<<<dimGrid, dimBlock>>>(d_inputData, args.size, d_outputData);
      break;
    }
    default:
      throw std::runtime_error("Invalid kernel number: " + std::to_string(args.kernel_number));
  }
  cudaCheck(hipGetLastError());
}

int main(int argc, char* argv[]) {
  Args args;
  hipDeviceProp_t deviceProp = getDeviceProperties(0, true);
  if (parse_args(argc, argv, args, deviceProp) != 0) {
    return 1;
  }

  const std::vector<int> inputData = generateInputData(args.size);
  const unsigned long long cpuResult = reduceCpu(inputData);
  unsigned long long gpuResult = 0;

  {
    Timer timer("reduce on the GPU");
    auto d_inputData = make_cuda_unique<int>(args.size);
    auto d_outputData = make_cuda_unique<unsigned long long>(1);
    cudaCheck(hipHostRegister((void*)inputData.data(), args.size * sizeof(int), hipHostRegisterDefault));
    cudaCheck(hipMemcpy(d_inputData.get(), inputData.data(), args.size * sizeof(int), hipMemcpyHostToDevice));

    {
      CudaEventRecorder recorder("reduce on the GPU");
      runKernel(d_inputData.get(), d_outputData.get(), args);
      cudaCheck(hipDeviceSynchronize());
    }

    cudaCheck(hipMemcpy(&gpuResult, d_outputData.get(), sizeof(unsigned long long), hipMemcpyDeviceToHost));
  }
  printf("GPU result: %llu\n", gpuResult);
  printf("CPU result: %llu\n", cpuResult);
  printf("Difference: %llu\n", gpuResult > cpuResult ? gpuResult - cpuResult : cpuResult - gpuResult);

  return 0;
}